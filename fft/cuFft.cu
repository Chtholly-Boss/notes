
// includes, system
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

// includes, project

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>

// Complex data type
typedef float2 Complex;
void cuFft(Complex *d,Complex *h);
#define N 8

int main(int argc,char** argv) {
  // ! Allocate Memory
  int mem_size = sizeof(Complex) * N;
  // Allocate Host Memory for signal
  Complex * h_signal = 
    reinterpret_cast<Complex *> (malloc(mem_size)); 
  // Initialize the memory for the signal
  for (unsigned int i = 0; i < N; ++i) {
  h_signal[i].x = i;
  //h_signal[i].x = rand() / static_cast<float>(RAND_MAX);
  h_signal[i].y = 0;
  }
  // Allocate device memory for signal
  Complex *d_signal;
  hipMalloc(reinterpret_cast<void **>(&d_signal), mem_size);
  // Copy host memory to device
  hipMemcpy(d_signal, h_signal, mem_size, hipMemcpyHostToDevice);
  // ! Kernel Call
  cuFft(d_signal,h_signal);
  // ! Result Check
  //cudaMemcpy(h_signal, d_signal, mem_size,cudaMemcpyDeviceToHost);
  for (unsigned int i = 0; i < N; ++i) {
    printf("The %dth element:(x: %f, y: %f)\n",i,h_signal[i].x,h_signal[i].y);
  }
  // ! Clean up Memory
  free(h_signal);
  hipFree(d_signal);

  return 0;
}

void cuFft(Complex *d,Complex *h) {
  printf("cuFFT is Running...\n");
  hipfftHandle plan;
  hipfftPlan1d(&plan,N,HIPFFT_C2C,1);

  printf("Transforming Signal Using cuFFT\n");
  hipfftExecC2C(plan, reinterpret_cast<hipfftComplex *>(d),
                              reinterpret_cast<hipfftComplex *>(h),
                              HIPFFT_FORWARD);
  hipfftDestroy(plan);
}